// RUN: %check_clang_tidy %s cuda-unsafe-api-call %t -- \
// RUN:   -config="{CheckOptions: \
// RUN:             [{key: cuda-unsafe-api-call.HandlerName, \
// RUN:               value: 'CUDA_HANDLER'}] \
// RUN:             }" \
// RUN:   -- -isystem %clang_tidy_headers -std=c++14
#include <cuda/hip/hip_runtime.h>

class DummyContainer {
 public:
  int* begin();
  int* end();
};

#define DUMMY_CUDA_HANDLER(stmt) stmt
#define CUDA_HANDLER(stmt) do {auto err = stmt;} while(0)
#define API_CALL() do {hipDeviceReset();} while(0)

void errorCheck();
void errorCheck(hipError_t error);

void bad() {
  API_CALL();
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
  // There isn't supposed to be a fix here since it's a macro call

  hipDeviceReset();
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
  // CHECK-FIXES:  {{^}}  CUDA_HANDLER(hipDeviceReset());{{$}}
  errorCheck();

  if (true)
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset());{{$}}

  while (true)
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset());{{$}}

  do
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset());{{$}}
  while(false);

  switch (0) {
    case 0:
      hipDeviceReset();
      // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
      // CHECK-FIXES:  {{^}}      CUDA_HANDLER(hipDeviceReset());{{$}}
  }

  for(
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset());{{$}}
    ;
    hipDeviceReset()
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset()){{$}}
  ) hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}  ) CUDA_HANDLER(hipDeviceReset());{{$}}

  for(int i : DummyContainer())
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset());{{$}}

  auto x = ({
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: Unchecked CUDA API call.
    // CHECK-FIXES:  {{^}}    CUDA_HANDLER(hipDeviceReset());{{$}}
    true;
  });
}

int good() {
  DUMMY_CUDA_HANDLER(hipDeviceReset());

  if (hipDeviceReset()) {
    return 0;
  }

  switch (hipDeviceReset()) {
    case hipErrorInvalidValue: return 1;
    case hipErrorOutOfMemory: return 2;
    default: return 3;
  }

  auto err = ({hipDeviceReset();});
  // NOTE: We don't check that `errorCheck()` actually handles the error; we just assume it does.
  errorCheck(hipDeviceReset());
}
