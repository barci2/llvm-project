// RUN: %check_clang_tidy %s cuda-unsafe-api-call %t -- \
// RUN:   -config="{CheckOptions: \
// RUN:             [{key: cuda-unsafe-api-call.HandlerName, \
// RUN:               value: 'cudaHandler'}, \
// RUN:              {key: cuda-unsafe-api-call.AcceptedHandlers, \
// RUN:               value: 'CUDA_HANDLER, DUMMY_CUDA_HANDLER, \
// RUN:                       alternative::cudaAlternativeHandler, \
// RUN:                       cudaOtherAlternativeHandler, bad::cudaBadHandler'}] \
// RUN:             }" \
// RUN:   -- -isystem %clang_tidy_headers -std=c++14
#include <cuda/hip/hip_runtime.h>

#define DUMMY_CUDA_HANDLER(stmt) stmt
#define CUDA_HANDLER(stmt) do {auto err = stmt;} while(0)
#define API_CALL() do {hipDeviceReset();} while(0)
#define HANDLED_API_CALL() do {int err2 = hipDeviceReset();} while(0)

void cudaHandler();
void cudaHandler(hipError_t error);
void badCudaHandler(hipError_t error);

namespace alternative {

void cudaAlternativeHandler(hipError_t error);

void cudaOtherAlternativeHandler(hipError_t error);

} // namespace alternative

void bad() {
  API_CALL();
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
  // There isn't supposed to be a fix here since it's a macro call

  HANDLED_API_CALL();
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
  // There isn't supposed to be a fix here since it's a macro call

  hipDeviceReset();
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
  // CHECK-FIXES:  {{^}}  cudaHandler(hipDeviceReset());{{$}}
  cudaHandler();

  if (true)
    hipDeviceReset();
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
    // CHECK-FIXES:  {{^}}    cudaHandler(hipDeviceReset());{{$}}

  badCudaHandler(hipDeviceReset());
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
  // There isn't supposed to be a fix here since the result value is not unused

  int err = hipDeviceReset();
  // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
  // There isn't supposed to be a fix here since the result value is not unused

  if (hipDeviceReset()) {
    // CHECK-MESSAGES: :[[@LINE-1]]:{{[0-9]+}}: warning: CUDA API call not checked properly.
    // There isn't supposed to be a fix here since the result value is not unused
    return;
  }

}

void good() {
  cudaHandler(hipDeviceReset());
  alternative::cudaAlternativeHandler(hipDeviceReset());
  alternative::cudaOtherAlternativeHandler(hipDeviceReset());
  CUDA_HANDLER(hipDeviceReset() + 1);
  DUMMY_CUDA_HANDLER(hipDeviceReset());
}
